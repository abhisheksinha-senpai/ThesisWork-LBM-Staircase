#include "utilities.h"
#include "lbm.h"
#include "boundary.h"

using namespace std;

unsigned int mem_size_props;
float *f0_gpu,*f1_gpu,*f2_gpu;
float *rho_gpu,*ux_gpu,*uy_gpu, *uz_gpu;
float *prop_gpu;
float *scalar_host;

void getDeviceInfo()
{
    double bytesPerMiB = 1024.0*1024.0;
    double bytesPerGiB = 1024.0*1024.0*1024.0;
    
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));
    
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
    
    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem,&gpu_total_mem));

    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n",deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n",deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n",deviceProp.totalGlobalMem/bytesPerMiB);
    printf("        free memory: %.1f MiB\n",gpu_free_mem/bytesPerMiB);
    return;
}

void AllocateMemory()
{
    double bytesPerMiB = 1024.0*1024.0;
    checkCudaErrors(hipMalloc((void**)&f0_gpu,mem_size_0dir));
    checkCudaErrors(hipMalloc((void**)&f1_gpu,mem_size_n0dir));
    checkCudaErrors(hipMalloc((void**)&f2_gpu,mem_size_n0dir));
    checkCudaErrors(hipMalloc((void**)&rho_gpu,mem_size_scalar));
    checkCudaErrors(hipMalloc((void**)&ux_gpu,mem_size_scalar));
    checkCudaErrors(hipMalloc((void**)&uy_gpu,mem_size_scalar));
    checkCudaErrors(hipMalloc((void**)&uz_gpu,mem_size_scalar));
    checkCudaErrors(hipMalloc((void**)&gpu_boundary,mem_size_bound));
    checkCudaErrors(hipMalloc((void**)&gpu_normals,mem_size_normal));
    mem_size_props = 0;//7*NX/nThreads*NY*sizeof(float);
    // checkCudaErrors(hipMalloc((void**)&prop_gpu,mem_size_props));

    scalar_host  = (float*) malloc(mem_size_scalar);
    cpu_boundary = (bool *)malloc(mem_size_bound);
    cpu_normals = (short *)malloc(mem_size_normals);
    if(scalar_host == NULL || cpu_boundary == NULL || cpu_normals == NULL )
    {
        fprintf(stderr,"Error: unable to allocate required host memory (%.1f MiB).\n",mem_size_scalar/bytesPerMiB);
        exit(-1);
    }
    else
    {
        unsigned int gpu_total = mem_size_0dir + 2*mem_size_n0dir + 4* mem_size_scalar + mem_size_props + mem_size_bound + mem_size_normals;
        printf("Allocated %.1f MiB memory in CPU and %.1f MiB in GPU\n",mem_size_scalar/bytesPerMiB, gpu_total/bytesPerMiB);
    }
}

void DeallocateMemory()
{
  
    // free all memory allocatd on the GPU and host
    checkCudaErrors(hipFree(f0_gpu));
    checkCudaErrors(hipFree(f1_gpu));
    checkCudaErrors(hipFree(f2_gpu));
    checkCudaErrors(hipFree(rho_gpu));
    checkCudaErrors(hipFree(ux_gpu));
    checkCudaErrors(hipFree(uy_gpu));
    checkCudaErrors(hipFree(uz_gpu));
    // checkCudaErrors(hipFree(prop_gpu));
    checkCudaErrors(hipFree(gpu_boundary));
    checkCudaErrors(hipFree(gpu_normals));
    free(scalar_host);
    free(cpu_boundary);
    free(cpu_normals);
    
    // release resources associated with the GPU device
    hipDeviceReset();
}