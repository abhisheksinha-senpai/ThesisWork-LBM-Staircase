#include "hip/hip_runtime.h"
#include "boundary.h"
#include "lbm.h"
#include "utilities.h"

using namespace std;

__host__ void cpu_field_Initialization()
{
    defineBoundary();
    checkCudaErrors(hipMemcpy(gpu_boundary, cpu_boundary, mem_size_bound,  hipMemcpyHostToDevice));
    // blocks in grid
    dim3  grid(NX/nThreads, NY, NZ);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_field_Initialization<<< grid, threads >>>(gpu_boundary, rho_gpu, ux_gpu, uy_gpu, uz_gpu);
    getLastCudaError("gpu_field_Initialization kernel error");
}

__global__ void gpu_field_Initialization(bool *boundary, float *rho, float *ux, float *uy, float *uz)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idz = threadIdx.z +  blockIdx.y * blockDim.z;

    unsigned int sidx = gpu_scalar_index(idx, idy, idz);
    rho[sidx] = (boundary[sidx] == true)?1:0;
    ux[sidx] = 0.0;
    uy[sidx] = 0.0;
    uz[sidx] = 0.0;
}

__host__ void defineBoundary()
{
    for(int k=0;k<NZ;k++)
    {
        // cout<<"first loop"<<endl;
        for(int i=0;i<NY;i++)
        {
            // cout<<"sec loop"<<endl;
            for(int j=0;j<NX;j++)
            {
                // cout<<"third loop"<<endl;
                if(j<NX/8)
                {
                    if(i<NY/4 || i>NY/2)
                    {
                        // cout<<"1     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = true;
                    }
                    else
                    {
                        // cout<<"2     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = false;
                    }
                }
                else if(j<NX/4)
                {
                    if(i>(NY-NY/4) || i<NY/4)
                    {
                        // cout<<"3     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = true;
                    }
                    else
                    {
                        // cout<<"4     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = false;
                    }
                }
                else if(j<(NX/2 - NX/4))
                {
                    if(i<NY/2)
                    {
                        // cout<<"5     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = true;
                    }
                    else
                    {
                        // cout<<"6     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = false;
                    }
                }
                else  if(j<(NX/2))
                {
                    if(i<(NY - NY/4))
                    {
                        // cout<<"7     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = true;
                    }
                    else
                    {
                        // cout<<"8     "<<k*NX*NY + i*NX +j<<endl;
                        cpu_boundary[k*NX*NY + i*NX +j] = false;
                    }
                }
                else
                {
                    // cout<<"9     "<<k*NX*NY + i*NX +j<<endl;
                    cpu_boundary[k*NX*NY + i*NX +j] = false;
                }            
            }
        }
    }
}